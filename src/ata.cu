#include "hip/hip_runtime.h"
#include "strassen.cu"

void printm(double* arr_, int m, int n) {
  double* arr = (double *)malloc(m * n * sizeof(double));
  hipMemcpy(arr, arr_, m * n * sizeof(double), hipMemcpyDeviceToHost);
  for (int i = 0; i < m; i++)
  {
   for (int j = 0; j < n; j++)
   {
      printf("%f ", arr[j + i * n]);
   }

   // Newline for new row
   printf("\n");
  }
  printf("\n");
}

void GPU_trans(double *A, double *C,
    int lda, int ldc,
    int XA, int YA) {
  double one = 1.0;
  double zero = 0.0;
  hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, YA, XA, &one, A, lda, &zero, A, lda, C, ldc);
}

void GPU_mul_t(double *A, double *B, double *C,
    int lda, int ldb, int ldc,
    int XA, int XB, int XC,
    int YA, int YB, int YC,
    double alpha, double beta) {
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, XB, YA, XA, &alpha, B, ldb, A, lda, &beta, C, ldc);
}

void GPU_ata(double *A, double *C, int M, int N) {
  double one = 1.0;
  double zero = 0.0;
#if CMAJOR
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, M, &one, A, M, A, M, &zero, C, M);
#else
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, M, &one, A, N, A, M, &zero, C, N);
#endif
}

/*
  lda, ldc is the width in actual memory.
  XA is the width for computation.
  A = XA x YA
  C = XC x YC
*/
void ata(double *A, double *C,
    int lda, int ldc,
    int XA, int XC,
    int YA, int YC,
    int depth) {

  int XA2 = XA / 2;
  int XC2 = XC / 2;

  int YA2 = YA / 2;
  int YC2 = YC / 2;

  double *W_1, *W_2;
  int lw1 = YA2;
  int lw2 = lw1;
  hipMalloc((void **)&W_1, lw1 * YA2 * sizeof(double));
  hipMalloc((void **)&W_2, lw2 * YA2 * sizeof(double));
  
  double *A12_t, *A22_t;
  int lda_t = YA2;
  hipMalloc((void **)&A12_t, lda_t * XA2 * sizeof(double));
  hipMalloc((void **)&A22_t, lda_t * XA2 * sizeof(double));

  int dXA = XA2;
  int dYA = YA2 * lda;

  int dXC = XC2;
  int dYC = YC2 * ldc;

  double *A11, *A12, *A21, *A22;
  double *C11, *C12, *C21, *C22;

  A11 = A;
  A12 = A + dXA;
  A21 = A + dYA;
  A22 = A + dXA + dYA;

  C11 = C;
  // C12 = C + dXC;
  C21 = C + dYC;
  C22 = C + dXC + dYC;

  /* cutoff criteria */
  bool stop = false;

#if 0
  int cutoff = 2048;
  float mm = cutoff / XB2;
  float nn = cutoff / YA2;
  float kk = cutoff / XA2;
  if ((mm + nn + kk) >= 3) {
      stop = true;
  }
#endif

  if (depth <= 1 || stop) {
    double* A_t;
    hipMalloc((void **)&A_t, YA * XA * sizeof(double));
    GPU_trans(A, A_t, lda, YA, XA, YA);
    strassen(A_t, A, C, YA, lda, ldc, YA, XA, XC, XA, YA, YC, 1);
  }
  else {
    ata(A11, W_1, lda, lw1, XA2, YA2, YA2, YA2, depth - 1);  // S1 = ata(A11)
    ata(A21, W_2, lda, lw2, XA2, XA2, YA2, YA2, depth - 1);  // S2 = ata(A21)
    GPU_add(W_1, W_2, C11, lw1, lw2, ldc, XA2, YA2, 1.0, 1.0);  // C11 = S1 + S2
    ata(A12, W_1, lda, lw1, XA2, XA2, YA2, YA2, depth - 1);  // S3 = ata(A12)
    ata(A22, W_2, lda, lw2, XA2, XA2, YA2, YA2, depth - 1);  // S4 = ata(A22)
    GPU_add(W_1, W_2, C22, lw1, lw2, ldc, XA2, YA2, 1.0,  1.0);  // C22 = S3 + S4
    GPU_trans(A12, A12_t, lda, YA, XA2, YA2);  // A12_t
    strassen(A12_t, A11, W_1, YA, lda, lw1, YA2, XA2, YA2, XA2, YA2, YA2, depth - 1);  // S5 = strassen(A12_t, A11)
    GPU_trans(A22, A22_t, lda, YA, XA2, YA2);  // A22_t
    strassen(A22_t, A21, W_2, YA, lda, lw2, YA2, XA2, YA2, XA2, YA2, YA2, depth - 1);  // S6 = strassen(A22_t, A21)
    GPU_add(W_1, W_2, C21, lw1, lw2, ldc, XA2, YA2, 1.0,  1.0);  // C21 = S5 + S6
  }
  hipFree(W_1);
  hipFree(W_2);
  hipFree(A12_t);
  hipFree(A22_t);

  /* dynamic peeling fix-up */
  // int pxa = XA % 2;
  // int pya = YA % 2;
  // int pxc = XC % 2;
  // int pyc = YC % 2;

  // int nxa = XA - pxa;
  // int nya = YA - pya;
  // int nxc = XC - pxc;
  // int nyc = YC - pyc;

  // double *a12, *a21;
  // double *c12, *c21;
  // int dxa = nxa;
  // int dya = nya * lda;
  // int dxc = nxc;
  // int dyc = nyc * ldc;

  // a12 = A + dxa;
  // a21 = A + dya;
  // // a22 = A + dxa + dya;
  // // b22 = B + dxb + dyb;
  // // c12 = C + dxc;
  // c21 = C + dyc;
  // // c22 = C + dxc + dyc;

  // /*
  //   A11 = nxa x nya
  //   a12 = pxa x nya
  //   a21 = nxa x pya
  //   a22 = pxa x pya
  //  */
  // GPU_mul_t(a21, A11, c21, lda, lda, ldc, nxa, YA, XC, pya, nxa, pyc, 1.0, 0.0);
  // GPU_mul(a12, a21, C11, lda, lda, ldc, pxa, YA,  XC, YA, pxa, YC, 1.0, 1.0);
}


int main (int argc, char **argv) {
  if(argc != 6) {
    printf("Usage: ata <M> <N> <iter> <check> <depth>\n");
    return -1;
  }

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int iter = atoi(argv[3]);
  int check = atoi(argv[4]);
  int depth = atoi(argv[5]);

  int sizeA = M * N;
  int sizeC = N * N;
  int memSizeA = sizeA * sizeof(double);
  int memSizeC = sizeC * sizeof(double);

  double *h_A = (double *)malloc(memSizeA);
  double *h_C = (double *)malloc(memSizeC);
  double *v_C = (double *)malloc(memSizeC);

  for (int i = 0; i < sizeA; i++) {
    h_A[i] = i;
  }

  // for (int i = 0; i < sizeA; i++) {
  //   h_A[i] = i % 3;
  // }
  for (int i = 0; i < sizeC; i++) {
    h_C[i] = 0.0;
    v_C[i] = 0.0;
  }

  double *d_A, *d_C;
  hipMalloc((void**)&d_A, memSizeA);
  hipMalloc((void**)&d_C, memSizeC);
  hipMemcpy(d_A, h_A, memSizeA, hipMemcpyHostToDevice);

  if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }

  CudaTimer ct;
  ct.start();
  for (int i = 0; i < iter; i++) {
    ata(d_A, d_C, N, N, N, N, M, N, depth);
  }
  ct.stop();

  double strassenTime = ct.value() / iter;
  hipMemcpy(h_C, d_C, memSizeC, hipMemcpyDeviceToHost);
  printm(d_A, M, N);
  printm(d_C, N, N);

#if 0
  ct.start();
  for (int i = 0; i < iter; i++) {
    GPU_ata(d_A, d_C, M, N);
  }
  ct.stop();

  double classicTime = ct.value() / iter;
  hipMemcpy(v_C, d_C, memSizeC, hipMemcpyDeviceToHost);

  double speedup = classicTime / strassenTime;
  printf ("%d %d %.2f %.2f %.2f\n", M, N, strassenTime, classicTime, speedup);
#endif

  if (check) {
    double absErr = 0.0;
    for (int i = 0; i < sizeC; i++) {
      absErr += abs(h_C[i] - v_C[i]);
    }
    if (absErr > 1) {
      printf("CHECK: Absolute error: %lf\n", absErr);
    }
  }

  free(h_A);
  free(h_C);
  free(v_C);
  hipFree(d_A);
  hipFree(d_C);

  if (hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS shutdown error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }
}
