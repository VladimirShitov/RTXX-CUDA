#include "hip/hip_runtime.h"
#include "ata.h"
#include "strassen.cu"

void GPU_T(double *A, double *C,
    int lda, int ldc,
    int XA, int YA) {
  double one = 1.0;
  double zero = 0.0;
  cublasGeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, XA, YA, &one, A, lda, &zero, C, ldc, C, ldc);
}

void GPU_AtB(double *A, double *B, double *C,
    int lda, int ldb, int ldc,
    int XA, int XB, int XC,
    int YA, int YB, int YC,
    double alpha, double beta) {
  cublasGemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, XB, YA, XA, &alpha, B, ldb, A, lda, &beta, C, ldc);
}

/*
  lda, ldc is the width in actual memory.
  XA, XC is the width for computation.
  Returns the lower triangular part of C.
  A = XA x YA
  C = XC x YC
*/
void ata(double *A, double *C,
    int lda, int ldc,
    int XA, int XC,
    int YA, int YC,
    int depth) {

  int XA2 = XA / 2;
  int XC2 = XC / 2;

  int YA2 = YA / 2;
  int YC2 = YC / 2;

  double *W_1, *W_2;
  int ldw = XC2;
  hipMalloc((void **)&W_1, ldw * YC2 * sizeof(double));
  hipMalloc((void **)&W_2, ldw * YC2 * sizeof(double));

  int dXA = XA2;
  int dYA = YA2 * lda;

  int dXC = XC2;
  int dYC = YC2 * ldc;

  double *A11, *A12, *A21, *A22;
  double *C11, *C21, *C22;

  A11 = A;
  A12 = A + dXA;
  A21 = A + dYA;
  A22 = A + dXA + dYA;

  C11 = C;
  // C12 = C + dXC;
  C21 = C + dYC;
  C22 = C + dXC + dYC;

  /* cutoff criteria */
  bool stop = false;

  int cutoff = 2048;
  float mm = cutoff / XA2;
  float nn = cutoff / YA2;
  if ((mm + nn) >= 2) {
      stop = true;
  }

  if (depth <= 1 || stop) {
    GPU_AtB(A11, A11, W_1, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S1 = ata(A11)
    GPU_AtB(A21, A21, W_2, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S2 = ata(A21)
    GPU_add(W_1, W_2, C11, ldw, ldw, ldc, XC2, YC2, 1.0, 1.0);                      // C11 = S1 + S2
    GPU_AtB(A12, A12, W_1, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S3 = ata(A12)
    GPU_AtB(A22, A22, W_2, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S4 = ata(A22)
    GPU_add(W_1, W_2, C22, ldw, ldw, ldc, XC2, YC2, 1.0,  1.0);                     // C22 = S3 + S4
    GPU_AtB(A12, A11, W_1, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S5 = strassen(A12_t, A11)
    GPU_AtB(A22, A21, W_2, lda, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, 1.0, 0.0);  // S6 = strassen(A22_t, A21)
    GPU_add(W_1, W_2, C21, ldw, ldw, ldc, XC2, YC2, 1.0,  1.0);                     // C21 = S5 + S6
  }
  else {
    double *A2t;
    int ldt = YA2;
    hipMalloc((void **)&A2t, ldt * XA2 * sizeof(double));

    ata(A11, W_1, lda, ldw, XA2, XC2, YA2, YC2, depth - 1);                           // S1 = ata(A11)
    ata(A21, W_2, lda, ldw, XA2, XC2, YA2, YC2, depth - 1);                           // S2 = ata(A21)
    GPU_add(W_1, W_2, C11, ldw, ldw, ldc, XC2, YC2, 1.0, 1.0);                        // C11 = S1 + S2
    ata(A12, W_1, lda, ldw, XA2, XC2, YA2, YC2, depth - 1);                           // S3 = ata(A12)
    ata(A22, W_2, lda, ldw, XA2, XC2, YA2, YC2, depth - 1);                           // S4 = ata(A22)
    GPU_add(W_1, W_2, C22, ldw, ldw, ldc, XC2, YC2, 1.0,  1.0);                       // C22 = S3 + S4
    GPU_T(A12, A2t, lda, ldt, YA2, XA2);                                              // A12t
    strassen(A2t, A11, W_1, ldt, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, depth - 1);  // S5 = strassen(A12t, A11)
    GPU_T(A22, A2t, lda, ldt, YA2, XA2);                                              // A22t
    strassen(A2t, A21, W_2, ldt, lda, ldw, YA2, XA2, XC2, XA2, YA2, YC2, depth - 1);  // S6 = strassen(A22t, A21)
    GPU_add(W_1, W_2, C21, ldw, ldw, ldc, XC2, YC2, 1.0,  1.0);                       // C21 = S5 + S6

    hipFree(A2t);
    hipFree(A2t);
  }
  hipFree(W_1);
  hipFree(W_2);

  /* dynamic peeling fix-up */
  int pxa = XA % 2;
  int pya = YA % 2;
  int pxc = XC % 2;
  int pyc = YC % 2;

  int nxa = XA - pxa;
  int nya = YA - pya;
  int nxc = XC - pxc;
  int nyc = YC - pyc;

  double *a12, *a21;
  double *c21;
  int dxa = nxa;
  int dya = nya * lda;
  // int dxc = nxc;
  int dyc = nyc * ldc;

  a12 = A + dxa;
  a21 = A + dya;
  // a22 = A + dxa + dya;
  // c12 = C + dxc;
  c21 = C + dyc;
  // c22 = C + dxc + dyc;

  /*
    A11 = nxa x nya
    a12 = pxa x nya
    a21 = nxa x pya
    a22 = pxa x pya
   */
  GPU_AtB(a12, A, c21, lda, lda, ldc, YA, XA, XC, pxa, YA, pyc, 1.0, 0.0);
  GPU_AtB(a21, a21, C11, lda, lda, ldc, pya, nxa, nxc, nxa, pya, nyc, 1.0, 1.0);
}

// void printm(double* arr, int m, int n) {
//   for (int i = 0; i < m; i++) {
//    for (int j = 0; j < n; j++) {
//       printf("%f ", arr[j + i * n]);
//    }
//    printf("\n");
//   }
//   printf("\n");
// }


int main (int argc, char **argv) {
  if(argc != 6) {
    printf("Usage: %s <M> <N> <iter> <check> <depth>\n", argv[0]);
    return -1;
  }

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int iter = atoi(argv[3]);
  int check = atoi(argv[4]);
  int depth = atoi(argv[5]);

  int sizeA = M * N;
  int sizeC = N * N;
  int memSizeA = sizeA * sizeof(double);
  int memSizeC = sizeC * sizeof(double);

  // double *h_A = (double *)malloc(memSizeA);
  double *h_C = (double *)malloc(memSizeC);
  double *v_C = (double *)malloc(memSizeC);

  double *d_A, *d_C;
  hipMalloc((void**)&d_A, memSizeA);
  hipMalloc((void**)&d_C, memSizeC);

  hiprandGenerator_t rng;

  if (hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! cuRAND initialization error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }

  hiprandSetPseudoRandomGeneratorSeed(rng, rand());
  hiprandGenerateUniform(rng, d_A, sizeA);
  // hipMemcpy(h_A, d_A, memSizeA, hipMemcpyDeviceToHost);
  // printm(h_A, M, N);

  if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! cuBLAS initialization error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }

  CudaTimer ct;
  ct.start();
  for (int i = 0; i < iter; i++) {
    ata(d_A, d_C, N, N, N, N, M, N, depth);
  }
  ct.stop();

  double ataTime = ct.value() / iter;
  hipMemcpy(h_C, d_C, memSizeC, hipMemcpyDeviceToHost);
  // printm(h_C, N, N);

  ct.start();
  for (int i = 0; i < iter; i++) {
    GPU_AtB(d_A, d_A, d_C, N, N, N, M, N, N, N, M, N, 1.0, 0.0);
  }
  ct.stop();

  double classicTime = ct.value() / iter;
  hipMemcpy(v_C, d_C, memSizeC, hipMemcpyDeviceToHost);
  // printm(v_C, N, N);

  double speedup = classicTime / ataTime;
  printf ("M: %d; N: %d; AtA time: %.2f; classic time %.2f; speedup: %.2f\n", M, N, ataTime, classicTime, speedup);

  if (check) {
    double absErr = 0.0;
    for (int i = 0; i < M; i++) {
      for (int j = 0; j <= i; j++) {
        absErr += abs(h_C[i * N + j] - v_C[i * N + j]);
      }
    }
    if (absErr > 1.0) {
      printf("CHECK: Absolute error: %lf\n", absErr);
    }
  }

  // free(h_A);
  free(h_C);
  free(v_C);
  hipFree(d_A);
  hipFree(d_C);

  if (hiprandDestroyGenerator(rng) != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! cuRAND shutdown error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }

  if (hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! cuBLAS shutdown error\n");
    fflush(NULL);
    return EXIT_FAILURE;
  }
}
